#include "hip/hip_runtime.h"
#ifndef _DELITE_CUDA_
#define _DELITE_CUDA_

#include "DeliteCuda.h"

using namespace std;

list<void*>* lastAlloc = new list<void*>();
queue<FreeItem>* freeList = new queue<FreeItem>();
map<void*,list<void*>*>* cudaMemoryMap = new map<void*,list<void*>*>();

void addEvent(hipStream_t fromStream, hipStream_t toStream) {
  hipEvent_t event;
  hipEventCreateWithFlags(&event, hipEventDisableTiming);
  hipEventRecord(event, fromStream);
  hipStreamWaitEvent(toStream, event, 0);
  hipEventDestroy(event);
}

hipEvent_t addHostEvent(hipStream_t stream) {
  hipEvent_t event;
  hipEventCreateWithFlags(&event, hipEventDisableTiming | hipEventBlockingSync);
  hipEventRecord(event, stream);
  return event;
}

void freeCudaMemory(FreeItem item) {
    list< pair<void*,bool> >::iterator iter;
    for (iter = item.keys->begin(); iter != item.keys->end(); iter++) {
        //cout << "object ref: " << (long) *iter << endl;
        if(cudaMemoryMap->find((*iter).first) != cudaMemoryMap->end()) {
        	list<void*>* freePtrList = cudaMemoryMap->find((*iter).first)->second;
       		list<void*>::iterator iter2;
        	for (iter2 = freePtrList->begin(); iter2 != freePtrList->end(); iter2++) {
            	void* freePtr = *iter2;
            	hipFree(freePtr);
            	//if (hipFree(freePtr) != hipSuccess)
            	//    cout << "bad free pointer: " << (long) freePtr << endl;
            	//else
                	//cout << "freed successfully: " << (long) freePtr << endl;
       		}
        	cudaMemoryMap->erase((*iter).first);
        	delete freePtrList;
        	//if(!((*iter).second)) free((*iter).first);
		}
    }
    delete item.keys;
}

void DeliteCudaMalloc(void** ptr, size_t size) {

  while (freeList->size() != 0) {
    FreeItem item = freeList->front();
    if (hipEventQuery(item.event) != hipSuccess) {
      break;
    }
    freeList->pop();
    hipEventDestroy(item.event);
    freeCudaMemory(item);
  }

  while (hipMalloc(ptr, size) != hipSuccess) {
    if (freeList->size() == 0) {
      cout << "FATAL: Insufficient device memory" << endl;
      exit(-1);
    }
    FreeItem item = freeList->front();
    freeList->pop();

    while (hipEventQuery(item.event) != hipSuccess) {
      hipEventSynchronize(item.event);
    }
    hipEventDestroy(item.event);
    freeCudaMemory(item);
  }
  lastAlloc->push_back(*ptr);
}

size_t cudaHeapSize = 1024*1204;
char* bufferStart = 0;
size_t bufferSize = 5368709120/4;
char* bufferEnd;
char* bufferCurrent;


/* Implementations for temporary memory management */
#define CUDAMEM_ALIGNMENT 64
char *tempCudaMemPtr;
size_t tempCudaMemOffset;
size_t tempCudaMemSize;

void tempCudaMemInit(double tempMemRate) {
  size_t free, total;
  hipMemGetInfo(&free, &total);
  tempCudaMemSize = total * tempMemRate;
  tempCudaMemOffset = 0;
  if(hipMalloc(&tempCudaMemPtr, tempCudaMemSize) != hipSuccess) {
    cout << "FATAL (tempCudaMemInit): Insufficient device memory for tempCudaMem" << endl;
    exit(-1);
  }
  //cout << "Free:" << free << endl;
  //cout << "Total:" << total << endl;
  //cout << "tempMemSize:" << tempCudaMemSize << endl;
}

void tempCudaMemReset(void) {
  tempCudaMemOffset = 0;
}

size_t tempCudaMemAvailable(void) {
  return (tempCudaMemSize - tempCudaMemOffset - CUDAMEM_ALIGNMENT);
}

void DeliteCudaMallocTemp(void** ptr, size_t size) {
  size_t alignedSize = CUDAMEM_ALIGNMENT * (1 + size / CUDAMEM_ALIGNMENT);
  if(tempCudaMemOffset + alignedSize > tempCudaMemSize) {
    cout << "FATAL(DeliteCudaMallocTemp): Insufficient device memory for tempCudaMem" << endl;
    exit(-1);
  }
  else {
    *ptr = tempCudaMemPtr + tempCudaMemOffset;
    tempCudaMemOffset += alignedSize;
  }
}

void hostInit() {
	hipHostAlloc(&bufferStart, bufferSize, hipHostMallocDefault);
	bufferEnd = bufferStart + bufferSize;
	bufferCurrent = bufferStart;
}

void DeliteCudaMallocHost(void** ptr, size_t size) {
	if (bufferStart == 0) hostInit();
	if ((bufferCurrent + size) > bufferEnd)
		bufferCurrent = bufferStart;
	*ptr = bufferCurrent;
	bufferCurrent += size;
}

void DeliteCudaMemcpyHtoDAsync(void* dptr, void* sptr, size_t size) {
	hipMemcpyAsync(dptr, sptr, size, hipMemcpyHostToDevice, h2dStream);
}

void DeliteCudaMemcpyDtoHAsync(void* dptr, void* sptr, size_t size) {
	hipMemcpyAsync(dptr, sptr, size, hipMemcpyDeviceToHost, d2hStream);
	hipStreamSynchronize(d2hStream);
}

void DeliteCudaMemcpyDtoDAsync(void *dptr, void* sptr, size_t size) {
	hipMemcpyAsync(dptr, sptr, size, hipMemcpyDeviceToDevice, kernelStream);
}

void DeliteCudaMemset(void *ptr, int value, size_t count) {
	hipMemset(ptr,value,count);
}

void DeliteCudaCheckError(void) {
    hipDeviceSynchronize();
    printf("DeliteCuda ERROR: %s\n", hipGetErrorString(hipGetLastError()));
}

// TODO: Remove this kernel from here by generate it 
__global__ void kernel_offset(int *key, int *idx, int *offset, int size) {

  int idxX = threadIdx.x + blockIdx.x*blockDim.x;

  if(idxX == 0) {
    offset[1] = 0;
  }
  else if(idxX < size) {
    int keyVal = key[idxX];
    int keyValPrev = key[idxX-1]; 
    if(keyVal != keyValPrev) {
      offset[keyVal+1] = idxX;
    }
  }
  if(idxX == size-1) {
    int keyVal = key[idxX];
    offset[0] = keyVal+1;
    offset[keyVal+2] = size;
  }
}

#endif
